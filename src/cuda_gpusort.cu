#include "hip/hip_runtime.h"
/*
 * cuda_gpusort.cu
 *
 * Device implementation of GpuSort
 * ----
 * Copyright 2011-2025 (C) KaiGai Kohei <kaigai@kaigai.gr.jp>
 * Copyright 2014-2025 (C) PG-Strom Developers Team
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the PostgreSQL License.
 */
#include "cuda_common.h"
#include "float2.h"

INLINE_FUNCTION(int)
__gpusort_comp_rawkey(kern_context *kcxt,
					  const kern_sortkey_desc *sdesc,
					  const kern_data_store *kds_final,
					  const kern_tupitem *titem_x,
					  const kern_tupitem *titem_y)
{
	const void *addr_x = kern_fetch_minimal_tuple_attr(kds_final, titem_x,
													   sdesc->src_anum);
	const void *addr_y = kern_fetch_minimal_tuple_attr(kds_final, titem_y,
													   sdesc->src_anum);

	if (addr_x && addr_y)
	{
		const xpu_datum_operators *key_ops = sdesc->key_ops;
		xpu_datum_t	   *datum_x;
		xpu_datum_t	   *datum_y;
		int				sz, comp;

		/*
		 * !!!workaround for a bug!!!
		 *
		 * A couple of identical alloca() calls below were unintentionally
		 * optimized by the compiler.
		 * Probably, compiler considered that alloca() will return same
		 * value for the identical argument, thus datum_x and datum_y will
		 * have same value.
		 *
		 * datum_x = (xpu_datum_t *)alloca(key_ops->xpu_type_sizeof);
		 * datum_y = (xpu_datum_t *)alloca(key_ops->xpu_type_sizeof);
		 *
		 * If alloca() would be an immutable function, it is a right assumption,
		 * however, alloca() modified the current stack frame and allocates
		 * a temporary buffer. So, datum_x and datum_y should be different
		 * pointers.
		 */
		sz = TYPEALIGN(16, key_ops->xpu_type_sizeof);
		datum_x = (xpu_datum_t *)alloca(2 * sz);
		datum_y = (xpu_datum_t *)((char *)datum_x + sz);
		if (key_ops->xpu_datum_heap_read(kcxt, addr_x, datum_x) &&
			key_ops->xpu_datum_heap_read(kcxt, addr_y, datum_y) &&
			key_ops->xpu_datum_comp(kcxt, &comp, datum_x, datum_y))
			return sdesc->order_asc ? comp : -comp;
	}
	else if (addr_x && !addr_y)
		return (sdesc->nulls_first ? 1 : -1);	/* X is NOT NULL, Y is NULL */
	else if (!addr_x && addr_y)
		return (sdesc->nulls_first ? -1 : 1);	/* X is NULL, Y is NOT NULL */
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_pminmax_int64(kern_context *kcxt,
							 const kern_sortkey_desc *sdesc,
							 const kern_data_store *kds_final,
							 const kern_tupitem *titem_x,
							 const kern_tupitem *titem_y)
{
	const kagg_state__pminmax_int64_packed *x = (const kagg_state__pminmax_int64_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_x, sdesc->src_anum);
	const kagg_state__pminmax_int64_packed *y = (const kagg_state__pminmax_int64_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_y, sdesc->src_anum);
	if (x && (x->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
	{
		if (y && (y->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
		{
			if (x->value < y->value)
				return (sdesc->order_asc ? -1 : 1);
			if (x->value > y->value)
				return (sdesc->order_asc ? 1 : -1);
			return 0;
		}
		else
			return (sdesc->nulls_first ? 1 : -1);	/* X is NOT NULL, Y is NULL */
	}
	else if (y && (y->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
		return (sdesc->nulls_first ? -1 : 1);		/* X is NULL, Y is NOT NULL */
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_pminmax_fp64(kern_context *kcxt,
							const kern_sortkey_desc *sdesc,
							const kern_data_store *kds_final,
							const kern_tupitem *titem_x,
							const kern_tupitem *titem_y)
{
	const kagg_state__pminmax_fp64_packed *x = (const kagg_state__pminmax_fp64_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_x, sdesc->src_anum);
	const kagg_state__pminmax_fp64_packed *y = (const kagg_state__pminmax_fp64_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_y, sdesc->src_anum);
	if (x && (x->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
	{
		if (y && (y->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
		{
			if (x->value < y->value)
				return (sdesc->order_asc ? -1 : 1);
			if (x->value > y->value)
				return (sdesc->order_asc ? 1 : -1);
			return 0;
		}
		else
			return (sdesc->nulls_first ? 1 : -1);	/* X is NOT NULL, Y is NULL */
	}
	else if (y && (y->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
		return (sdesc->nulls_first ? -1 : 1);		/* X is NULL, Y is NOT NULL */
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_psum_int64(kern_context *kcxt,
						  const kern_sortkey_desc *sdesc,
						  const kern_data_store *kds_final,
						  const kern_tupitem *titem_x,
						  const kern_tupitem *titem_y)
{
	const kagg_state__psum_int_packed *x = (const kagg_state__psum_int_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_x, sdesc->src_anum);
	const kagg_state__psum_int_packed *y = (const kagg_state__psum_int_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_y, sdesc->src_anum);
	if (x && x->nitems > 0)
	{
		if (y && y->nitems > 0)
		{
			if (x->sum < y->sum)
				return (sdesc->order_asc ? -1 : 1);
			if (x->sum > y->sum)
				return (sdesc->order_asc ? 1 : -1);
			return 0;
		}
		else
			return (sdesc->nulls_first ? 1 : -1);	/* X!=NULL and Y==NULL */
	}
	else if (y && y->nitems > 0)
		return (sdesc->nulls_first ? -1 : 1);		/* X==NULL and Y!=NULL */
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_psum_fp64(kern_context *kcxt,
						  const kern_sortkey_desc *sdesc,
						  const kern_data_store *kds_final,
						  const kern_tupitem *titem_x,
						  const kern_tupitem *titem_y)
{
	const kagg_state__psum_fp_packed *x = (const kagg_state__psum_fp_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_x, sdesc->src_anum);
	const kagg_state__psum_fp_packed *y = (const kagg_state__psum_fp_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_y, sdesc->src_anum);
	if (x && x->nitems > 0)
	{
		if (y && y->nitems > 0)
		{
			if (x->sum < y->sum)
				return (sdesc->order_asc ? -1 : 1);
			if (x->sum > y->sum)
				return (sdesc->order_asc ? 1 : -1);
			return 0;
		}
		else
			return (sdesc->nulls_first ? 1 : -1);	/* X!=NULL and Y==NULL */
	}
	else if (y && y->nitems > 0)
		return (sdesc->nulls_first ? -1 : 1);		/* X==NULL and Y!=NULL */
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_psum_numeric(kern_context *kcxt,
							const kern_sortkey_desc *sdesc,
							const kern_data_store *kds_final,
							const kern_tupitem *titem_x,
							const kern_tupitem *titem_y)
{
	const kagg_state__psum_numeric_packed *x = (const kagg_state__psum_numeric_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_x, sdesc->src_anum);
	const kagg_state__psum_numeric_packed *y = (const kagg_state__psum_numeric_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem_y, sdesc->src_anum);
	if (x && x->nitems > 0)
	{
		if (y && y->nitems > 0)
		{
			xpu_numeric_t x_datum;
			xpu_numeric_t y_datum;
			int		xspecial = (x->attrs & __PAGG_NUMERIC_ATTRS__MASK);
			int		yspecial = (y->attrs & __PAGG_NUMERIC_ATTRS__MASK);
			int		comp;

			if (xspecial == 0)
			{
				x_datum.kind = XPU_NUMERIC_KIND__VALID;
				x_datum.weight = (int16_t)(x->attrs & __PAGG_NUMERIC_ATTRS__WEIGHT);
				x_datum.u.value = __fetch_int128_packed(&x->sum);
			}
			else if (xspecial == __PAGG_NUMERIC_ATTRS__PINF)
				x_datum.kind = XPU_NUMERIC_KIND__POS_INF;
			else if (xspecial == __PAGG_NUMERIC_ATTRS__NINF)
				x_datum.kind = XPU_NUMERIC_KIND__NEG_INF;
			else
				x_datum.kind = XPU_NUMERIC_KIND__NAN;
			x_datum.expr_ops = &xpu_numeric_ops;

			if (yspecial == 0)
			{
				y_datum.kind = XPU_NUMERIC_KIND__VALID;
				y_datum.weight = (int16_t)(y->attrs & __PAGG_NUMERIC_ATTRS__WEIGHT);
				y_datum.u.value = __fetch_int128_packed(&y->sum);
			}
			else if (yspecial == __PAGG_NUMERIC_ATTRS__PINF)
				y_datum.kind = XPU_NUMERIC_KIND__POS_INF;
			else if (yspecial == __PAGG_NUMERIC_ATTRS__NINF)
				y_datum.kind = XPU_NUMERIC_KIND__NEG_INF;
			else
				y_datum.kind = XPU_NUMERIC_KIND__NAN;
			y_datum.expr_ops = &xpu_numeric_ops;

			sdesc->key_ops->xpu_datum_comp(kcxt,
										   &comp,
										   (xpu_datum_t *)&x_datum,
										   (xpu_datum_t *)&y_datum);
			return (sdesc->order_asc ? comp : -comp);
		}
		else
			return (sdesc->nulls_first ? 1 : -1)	/* X!=NULL, Y==NULL */;
	}
	else if (y && y->nitems > 0)
		return (sdesc->nulls_first ? -1 : 1);		/* X==NULL, Y!=NULL */
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_precomp_fp64(const kern_sortkey_desc *sdesc,
							const kern_tupitem *titem_x,
							const kern_tupitem *titem_y)
{
	const char *addr_x = ((char *)titem_x + titem_x->t_len + sdesc->buf_offset);
	const char *addr_y = ((char *)titem_y + titem_y->t_len + sdesc->buf_offset);
	bool		notnull_x = *addr_x++;
	bool		notnull_y = *addr_y++;
	float8_t	fval_x;
	float8_t	fval_y;

	if (notnull_x && notnull_y)
	{
		memcpy(&fval_x, addr_x, sizeof(float8_t));
		memcpy(&fval_y, addr_y, sizeof(float8_t));
		if (fval_x < fval_y)
			return (sdesc->order_asc ? -1 : 1);
		if (fval_x > fval_y)
			return (sdesc->order_asc ? 1 : -1);
	}
	else if (notnull_x && !notnull_y)
		return (sdesc->nulls_first ? 1 : -1);
	else if (!notnull_x && notnull_y)
		return (sdesc->nulls_first ? -1 : 1);
	return 0;
}

INLINE_FUNCTION(int)
__gpusort_comp_keys(kern_context *kcxt,
					const kern_expression *sort_kexp,
					const kern_data_store *kds_final,
					const kern_tupitem *titem_x,
					const kern_tupitem *titem_y)
{
	if (!titem_x)
		return (!titem_y ? 0 : 1);
	else if (!titem_y)
		return -1;

	for (int k=0; k < sort_kexp->u.sort.nkeys; k++)
	{
		const kern_sortkey_desc *sdesc = &sort_kexp->u.sort.desc[k];
		int		comp;

		switch (sdesc->kind)
		{
			case KSORT_KEY_KIND__VREF:
				comp = __gpusort_comp_rawkey(kcxt, sdesc, kds_final, titem_x, titem_y);
				break;
			case KSORT_KEY_KIND__PMINMAX_INT64:
				comp = __gpusort_comp_pminmax_int64(kcxt, sdesc, kds_final, titem_x, titem_y);
				break;
			case KSORT_KEY_KIND__PMINMAX_FP64:
				comp = __gpusort_comp_pminmax_fp64(kcxt, sdesc, kds_final, titem_x, titem_y);
				break;
			case KSORT_KEY_KIND__PSUM_INT64:
				comp = __gpusort_comp_psum_int64(kcxt, sdesc, kds_final, titem_x, titem_y);
				break;
			case KSORT_KEY_KIND__PSUM_FP64:
				comp = __gpusort_comp_psum_fp64(kcxt, sdesc, kds_final, titem_x, titem_y);
				break;
			case KSORT_KEY_KIND__PSUM_NUMERIC:
				comp = __gpusort_comp_psum_numeric(kcxt, sdesc, kds_final, titem_x, titem_y);
				break;
			case KSORT_KEY_KIND__PAVG_INT64:
			case KSORT_KEY_KIND__PAVG_FP64:
			case KSORT_KEY_KIND__PAVG_NUMERIC:
			case KSORT_KEY_KIND__PVARIANCE_SAMP:
			case KSORT_KEY_KIND__PVARIANCE_POP:
			case KSORT_KEY_KIND__PCOVAR_CORR:
			case KSORT_KEY_KIND__PCOVAR_SAMP:
			case KSORT_KEY_KIND__PCOVAR_POP:
			case KSORT_KEY_KIND__PCOVAR_AVGX:
			case KSORT_KEY_KIND__PCOVAR_AVGY:
			case KSORT_KEY_KIND__PCOVAR_COUNT:
			case KSORT_KEY_KIND__PCOVAR_INTERCEPT:
			case KSORT_KEY_KIND__PCOVAR_REGR_R2:
			case KSORT_KEY_KIND__PCOVAR_REGR_SLOPE:
			case KSORT_KEY_KIND__PCOVAR_REGR_SXX:
			case KSORT_KEY_KIND__PCOVAR_REGR_SXY:
			case KSORT_KEY_KIND__PCOVAR_REGR_SYY:
				/* pre-computed float8 values */
				comp = __gpusort_comp_precomp_fp64(sdesc, titem_x, titem_y);
				break;
			default:
				/* Bug? should not happen */
				comp = 0;
				break;
		}
		if (comp != 0)
			return comp;
	}
	return 0;
}

/*
 * kern_gpusort_exec_bitonic
 */
KERNEL_FUNCTION(void)
kern_gpusort_exec_bitonic(kern_session_info *session,
						  kern_gputask *kgtask,
						  kern_data_store *kds_final,
						  uint32_t	nr_threads,
						  uint64_t *row_index,
						  int scale, int step)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	const char	   *end = (const char *)kds_final + kds_final->length;
	kern_context   *kcxt;
	uint32_t		thread_id;

	/* sanity checks */
	assert(get_local_size() <= CUDA_MAXTHREADS_PER_BLOCK);
	assert((nr_threads & (nr_threads-1)) == 0);
	/* save the GPU-Task specific read-only properties */
	if (get_local_id() == 0)
	{
		stromTaskProp__cuda_dindex        = kgtask->cuda_dindex;
		stromTaskProp__cuda_stack_limit   = kgtask->cuda_stack_limit;
		stromTaskProp__partition_divisor  = kgtask->partition_divisor;
		stromTaskProp__partition_reminder = kgtask->partition_reminder;
	}
	/* setup execution context */
	INIT_KERNEL_CONTEXT(kcxt, session, NULL);
	if (!row_index)
		row_index = KDS_GET_ROWINDEX(kds_final);
	for (thread_id = get_global_id();
		 thread_id < nr_threads;
		 thread_id += get_global_size())
	{
		uint32_t	base = ((thread_id >> scale) << (scale+1));
		uint32_t	m_bits = (thread_id & ((1U<<scale)-1)) >> step;
        uint32_t	l_bits = (thread_id & ((1U<<step)-1));
        uint32_t	index = base + (m_bits << (step+1)) + l_bits;
        uint32_t	buddy = index + (1U << step);
		bool		direction = (thread_id & (1U<<scale));
		int			comp;
		kern_tupitem *titem_x = NULL;
		kern_tupitem *titem_y = NULL;

		if (row_index[index] != 0)
			titem_x = (kern_tupitem *)(end - row_index[index]);
		if (row_index[buddy] != 0)
			titem_y = (kern_tupitem *)(end - row_index[buddy]);
		comp = __gpusort_comp_keys(kcxt, sort_kexp, kds_final, titem_x, titem_y);
		if (direction ? comp < 0 : comp > 0)
		{
			uint64_t	temp = row_index[index];

			row_index[index] = row_index[buddy];
			row_index[buddy] = temp;
		}
	}
}

INLINE_FUNCTION(void)
__gpusort_prep_pavg_int64(kern_context *kcxt,
						  const kern_data_store *kds_final,
						  const kern_tupitem *titem,
						  const kern_sortkey_desc *sdesc)
{
	const void *addr;
	char	   *dest;

	dest = ((char *)titem + titem->t_len + sdesc->buf_offset);
	addr = kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	if (!addr)
		*dest++ = false;
	else
	{
		const kagg_state__psum_int_packed *r =
			(const kagg_state__psum_int_packed *)addr;

		if (r->nitems == 0)
			*dest++ = false;
		else
		{
			double	fval = (double)r->sum / (double)r->nitems;

			*dest++ = true;
			memcpy(dest, &fval, sizeof(double));
		}
	}
}

INLINE_FUNCTION(void)
__gpusort_prep_pavg_fp64(kern_context *kcxt,
						 const kern_data_store *kds_final,
						 const kern_tupitem *titem,
						 const kern_sortkey_desc *sdesc)
{
	const void *addr;
	char	   *dest;

	dest = ((char *)titem + titem->t_len + sdesc->buf_offset);
	addr = kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	if (!addr)
		*dest++ = false;
	else
	{
		const kagg_state__psum_fp_packed *r =
			(const kagg_state__psum_fp_packed *)addr;

		if (r->nitems == 0)
			*dest++ = false;
		else
		{
			double	fval = r->sum / (double)r->nitems;

			*dest++ = true;
			memcpy(dest, &fval, sizeof(double));
		}
	}
}

INLINE_FUNCTION(void)
__gpusort_prep_pavg_numeric(kern_context *kcxt,
							const kern_data_store *kds_final,
							const kern_tupitem *titem,
							const kern_sortkey_desc *sdesc)
{
	const void *addr;
	char	   *dest;

	dest = ((char *)titem + titem->t_len + sdesc->buf_offset);
	addr = kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	if (!addr)
		*dest++ = false;
	else
	{
		const kagg_state__psum_numeric_packed *r =
            (const kagg_state__psum_numeric_packed *)addr;

		if (r->nitems == 0)
			*dest++ = false;
		else
		{
			uint32_t	special = (r->attrs & __PAGG_NUMERIC_ATTRS__MASK);
			int16_t		weight  = (r->attrs & __PAGG_NUMERIC_ATTRS__WEIGHT);
			float8_t	fval;

			*dest++ = true;
			if (special == 0)
			{
				int128_t	x, rem = __fetch_int128_packed(&r->sum);
				int64_t		div = r->nitems;
				float8_t	base = 1.0;
				float8_t	prev;
				bool		negative = false;

				if (rem < 0)
				{
					rem = -rem;
					negative = true;
				}
				/* integer portion */
				x = rem / div;
				fval = (double)x;
				rem -= x * div;

				while (rem != 0)
				{
					base /= 2.0;
					rem *= 2;
					if (rem > div)
					{
						prev = fval;
						fval += base;
						if (fval == prev)
							break;
						rem -= div;
					}
				}
				while (weight < 0)
				{
					fval *= 10.0;
					weight++;
				}
				while (weight > 0)
				{
					fval /= 10.0;
					weight--;
				}
				if (negative)
					fval = -fval;
			}
			else if (special == __PAGG_NUMERIC_ATTRS__PINF)
				fval = INFINITY;
			else if (special == __PAGG_NUMERIC_ATTRS__NINF)
				fval = -INFINITY;
			else
				fval = NAN;
			memcpy(dest, &fval, sizeof(float8_t));
		}
	}
}

INLINE_FUNCTION(void)
__gpusort_prep_pvariance(kern_context *kcxt,
						 const kern_data_store *kds_final,
						 const kern_tupitem *titem,
						 const kern_sortkey_desc *sdesc)
{
	const void *addr;
	char	   *dest;

	dest = ((char *)titem + titem->t_len + sdesc->buf_offset);
	addr = kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	if (!addr)
		*dest++ = false;
	else
	{
		const kagg_state__stddev_packed *r =
			(const kagg_state__stddev_packed *)addr;

		if (r->nitems == 0)
			*dest++ = false;
		else
		{
			float8_t	fval = 0.0;
			bool		isnull = false;

			switch (sdesc->kind)
			{
				case KSORT_KEY_KIND__PVARIANCE_SAMP:
					if (r->nitems < 2)
						isnull = true;
					else
					{
						double	N = (double)r->nitems;
						fval = (N * r->sum_x2 - r->sum_x * r->sum_x) / (N * (N - 1.0));
					}
				case KSORT_KEY_KIND__PVARIANCE_POP:
					if (r->nitems < 1)
						isnull = true;
					else
					{
						double	N = (double)r->nitems;
						fval = (N * r->sum_x2 - r->sum_x * r->sum_x) / (N * N);
					}
					break;
				default:
					isnull = true;
					break;
			}
			if (isnull)
				*dest++ = false;
			else
			{
				*dest++ = true;
				memcpy(dest, &fval, sizeof(float8_t));
			}
		}
	}
}

INLINE_FUNCTION(void)
__gpusort_prep_pcovariance(kern_context *kcxt,
						   const kern_data_store *kds_final,
						   const kern_tupitem *titem,
						   const kern_sortkey_desc *sdesc)
{
	const void *addr;
	char	   *dest;

	dest = ((char *)titem + titem->t_len + sdesc->buf_offset);
	addr = kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	if (!addr)
		*dest++ = false;
	else
	{
		const kagg_state__covar_packed *r =
			(const kagg_state__covar_packed *)addr;

		if (r->nitems == 0)
			*dest++ = false;
		else
		{
			float8_t	fval = 0.0;
			bool		isnull = false;

			switch (sdesc->kind)
			{
				case KSORT_KEY_KIND__PCOVAR_CORR:
					if (r->nitems < 1 ||
						r->sum_xx == 0.0 ||
						r->sum_yy == 0.0)
						isnull = true;
					else
						fval = r->sum_xy / sqrt(r->sum_xx * r->sum_yy);
					break;
				case KSORT_KEY_KIND__PCOVAR_SAMP:
					if (r->nitems < 2)
						isnull = true;
					else
						fval = r->sum_xy / (double)(r->nitems - 1);
					break;
                case KSORT_KEY_KIND__PCOVAR_POP:
					if (r->nitems < 1)
						isnull = true;
					else
						fval = r->sum_xy / (double)r->nitems;
					break;
                case KSORT_KEY_KIND__PCOVAR_AVGX:
					if (r->nitems < 1)
						isnull = true;
					else
						fval = r->sum_x / (double)r->nitems;
					break;
                case KSORT_KEY_KIND__PCOVAR_AVGY:
					if (r->nitems < 1)
						isnull = true;
					else
						fval = r->sum_x / (double)r->nitems;
					break;
                case KSORT_KEY_KIND__PCOVAR_COUNT:
					fval = (double)r->nitems;
					break;
                case KSORT_KEY_KIND__PCOVAR_INTERCEPT:
					if (r->nitems < 1 || r->sum_xx == 0.0)
						isnull = true;
					else
						fval = (r->sum_y -
								r->sum_x * r->sum_xy / r->sum_xx) / (double)r->nitems;
					break;
                case KSORT_KEY_KIND__PCOVAR_REGR_R2:
					if (r->nitems < 1 || r->sum_xx == 0.0 || r->sum_yy == 0.0)
						isnull = true;
					else
						fval = (r->sum_xy * r->sum_xy) / (r->sum_xx * r->sum_yy);
					break;
                case KSORT_KEY_KIND__PCOVAR_REGR_SLOPE:
					if (r->nitems < 1 || r->sum_xx == 0.0)
						isnull = true;
					else
						fval = (r->sum_xy / r->sum_xx);
					break;
				case KSORT_KEY_KIND__PCOVAR_REGR_SXX:
					if (r->nitems < 1)
						isnull = true;
					else
						fval = r->sum_xx;
					break;
				case KSORT_KEY_KIND__PCOVAR_REGR_SXY:
					if (r->nitems < 1)
						isnull = true;
					else
						fval = r->sum_xy;
					break;
                case KSORT_KEY_KIND__PCOVAR_REGR_SYY:
					if (r->nitems < 1)
						isnull = true;
					else
						fval = r->sum_yy;
					break;
				default:
					isnull = true;
					break;
			}

			if (isnull)
				*dest++ = false;
			else
			{
				*dest++ = true;
				memcpy(dest, &fval, sizeof(float8_t));
			}
		}
	}
}

/*
 * per-tuple preparation on demand
 */
INLINE_FUNCTION(void)
__gpusort_prep_tupitem(kern_context *kcxt,
					   const kern_expression *sort_kexp,
					   const kern_data_store *kds_final,
					   uint32_t kds_index)
{
	const kern_tupitem *titem = KDS_GET_TUPITEM(kds_final, kds_index);

	for (int k=0; k < sort_kexp->u.sort.nkeys; k++)
	{
		const kern_sortkey_desc *sdesc = &sort_kexp->u.sort.desc[k];

		switch (sdesc->kind)
		{
			case KSORT_KEY_KIND__PAVG_INT64:
				__gpusort_prep_pavg_int64(kcxt, kds_final, titem, sdesc);
				break;
			case KSORT_KEY_KIND__PAVG_FP64:
				__gpusort_prep_pavg_fp64(kcxt, kds_final, titem, sdesc);
				break;
			case KSORT_KEY_KIND__PAVG_NUMERIC:
				__gpusort_prep_pavg_numeric(kcxt, kds_final, titem, sdesc);
				break;
			case KSORT_KEY_KIND__PVARIANCE_SAMP:
			case KSORT_KEY_KIND__PVARIANCE_POP:
				__gpusort_prep_pvariance(kcxt, kds_final, titem, sdesc);
				break;
			case KSORT_KEY_KIND__PCOVAR_CORR:
			case KSORT_KEY_KIND__PCOVAR_SAMP:
			case KSORT_KEY_KIND__PCOVAR_POP:
			case KSORT_KEY_KIND__PCOVAR_AVGX:
			case KSORT_KEY_KIND__PCOVAR_AVGY:
			case KSORT_KEY_KIND__PCOVAR_COUNT:
			case KSORT_KEY_KIND__PCOVAR_INTERCEPT:
			case KSORT_KEY_KIND__PCOVAR_REGR_R2:
			case KSORT_KEY_KIND__PCOVAR_REGR_SLOPE:
			case KSORT_KEY_KIND__PCOVAR_REGR_SXX:
			case KSORT_KEY_KIND__PCOVAR_REGR_SXY:
			case KSORT_KEY_KIND__PCOVAR_REGR_SYY:
				__gpusort_prep_pcovariance(kcxt, kds_final, titem, sdesc);
				break;
			default:
				/* nothing to do */
				break;
		}
	}
}

/*
 * kern_gpusort_prep_buffer
 */
KERNEL_FUNCTION(void)
kern_gpusort_prep_buffer(kern_session_info *session,
						 kern_gputask *kgtask,
						 kern_data_store *kds_final,
						 uint32_t nr_threads,
						 uint64_t *row_index)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	kern_context   *kcxt;
	uint32_t		nrooms = 2 * nr_threads;
	uint32_t		index;

	/* sanity checks */
	assert(get_local_size() <= CUDA_MAXTHREADS_PER_BLOCK);
	assert(kds_final->nitems >= nr_threads &&
		   kds_final->nitems <= nrooms);
	/* save the GPU-Task specific read-only properties */
	if (get_local_id() == 0)
	{
		stromTaskProp__cuda_dindex        = kgtask->cuda_dindex;
		stromTaskProp__cuda_stack_limit   = kgtask->cuda_stack_limit;
		stromTaskProp__partition_divisor  = kgtask->partition_divisor;
		stromTaskProp__partition_reminder = kgtask->partition_reminder;
	}
	/* setup execution context */
	INIT_KERNEL_CONTEXT(kcxt, session, NULL);
	for (index=get_global_id(); index < nrooms; index += get_global_size())
	{
		if (index < kds_final->nitems)
		{
			if (sort_kexp->u.sort.needs_finalization)
				__gpusort_prep_tupitem(kcxt, sort_kexp, kds_final, index);
			if (row_index)
				row_index[index] = KDS_GET_ROWINDEX(kds_final)[index];
		}
		else if (row_index)
			row_index[index] = NULL;
		else
			KDS_GET_ROWINDEX(kds_final)[index] = NULL;
	}
}

/*
 * kern_windowrank_exec_row_number
 */
KERNEL_FUNCTION(void)
kern_windowrank_exec_row_number(kern_session_info *session,
								kern_data_store *kds_final,
								uint32_t *partition_hash_array,
								uint64_t *windowrank_row_index)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	uint32_t   *orderby_hash_array = partition_hash_array + kds_final->nitems;
	uint32_t   *results_array = orderby_hash_array + kds_final->nitems;
	uint32_t	index;

	assert(sort_kexp->u.sort.window_rank_func == KSORT_WINDOW_FUNC__ROW_NUMBER);
	for (index = get_global_id();
		 index < kds_final->nitems;
		 index += get_global_size())
	{
		uint32_t	start = 0;
		uint32_t	end = index;
		uint32_t	my_hash;

		my_hash = partition_hash_array[index];
		while (start != end)
		{
			uint32_t	curr = (start + end) / 2;

			if (partition_hash_array[curr] == my_hash)
				end = curr;
			else
				start = curr + 1;
		}
		assert(partition_hash_array[start] == my_hash);
		if (index - start < sort_kexp->u.sort.window_rank_limit - 1)
		{
			results_array[index] = 1;
			windowrank_row_index[index] = KDS_GET_ROWINDEX(kds_final)[index];
		}
		else
		{
			results_array[index] = 0;
			windowrank_row_index[index] = 0UL;
		}
	}
}

/*
 * kern_windowrank_exec_rank
 */
KERNEL_FUNCTION(void)
kern_windowrank_exec_rank(kern_session_info *session,
						  kern_data_store *kds_final,
						  uint32_t *partition_hash_array,
						  uint64_t *windowrank_row_index)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	uint32_t   *orderby_hash_array = partition_hash_array + kds_final->nitems;
	uint32_t   *results_array = orderby_hash_array + kds_final->nitems;
	uint32_t	index;

	assert(sort_kexp->u.sort.window_rank_func == KSORT_WINDOW_FUNC__RANK);
	for (index = get_global_id();
		 index < kds_final->nitems;
		 index += get_global_size())
	{
		uint32_t	start = 0;
		uint32_t	end = index;
		uint32_t	my_hash;
		uint32_t	part_leader;

		my_hash = partition_hash_array[index];
		while (start != end)
		{
			uint32_t	curr = (start + end) / 2;

			if (partition_hash_array[curr] == my_hash)
				end = curr;
			else
				start = curr + 1;
		}
		assert(partition_hash_array[start] == my_hash);
		part_leader = start;
		end = index;
		my_hash = orderby_hash_array[index];
		while (start != end)
		{
			uint32_t	curr = (start + end) / 2;

			if (orderby_hash_array[curr] == my_hash)
				end = curr;
			else
				start = curr + 1;
		}
		assert(orderby_hash_array[start] == my_hash);
		assert(part_leader <= start && start <= index);
		if (start - part_leader < sort_kexp->u.sort.window_rank_limit - 1)
		{
			//printf("RANK-FOUND (%u %u %u) delta=%u %u\n", part_leader, start, index, start - part_leader, index - start);
			results_array[index] = 1;
			windowrank_row_index[index] = KDS_GET_ROWINDEX(kds_final)[index];
		}
		else
		{
			results_array[index] = 0;
			windowrank_row_index[index] = 0UL;
		}
	}
}

/*
 * kern_windowrank_exec_dense_rank
 */
KERNEL_FUNCTION(void)
kern_windowrank_exec_dense_rank(kern_session_info *session,
								kern_data_store *kds_final,
								uint32_t *partition_hash_array,
								uint64_t *windowrank_row_index,
								int phase)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	uint32_t   *orderby_hash_array = partition_hash_array + kds_final->nitems;
	uint32_t   *results_array = orderby_hash_array + kds_final->nitems;
	uint32_t	index;

	assert(sort_kexp->u.sort.window_rank_func == KSORT_WINDOW_FUNC__DENSE_RANK);
	for (index = get_global_id();
		 index < kds_final->nitems;
		 index += get_global_size())
	{
		if (phase == 0)
		{
			if (index == 0 ||
				partition_hash_array[index] != partition_hash_array[index-1] ||
				orderby_hash_array[index] != orderby_hash_array[index-1])
			{
				results_array[index] = 1;
			}
			else
			{
				results_array[index] = 0;
			}
		}
		else if (phase == 1)
		{
			uint32_t	start = 0;
			uint32_t	end = index;
			uint32_t	my_hash = partition_hash_array[index];;

			while (start != end)
			{
				uint32_t	curr = (start + end) / 2;

				if (partition_hash_array[curr] == my_hash)
					end = curr;
				else
					start = curr + 1;
			}
			assert(start <= index);
			assert(partition_hash_array[start] == my_hash);
			if (results_array[index] -
				results_array[start] < sort_kexp->u.sort.window_rank_limit - 1)
			{
				windowrank_row_index[index] = KDS_GET_ROWINDEX(kds_final)[index];
			}
			else
			{
				windowrank_row_index[index] = 0UL;
			}
		}
		else if (phase == 2)
		{
			results_array[index] = (windowrank_row_index[index] != 0UL);
		}
		else
		{
			break;		/* should not happen */
		}
	}
}

/*
 * internal APIs to load sorting keys
 */
INLINE_FUNCTION(bool)
__gpusort_load_rawkey(kern_context *kcxt,
					  const kern_sortkey_desc *sdesc,
					  const kern_data_store *kds_final,
					  const kern_tupitem *titem,
					  xpu_datum_t *xdatum)
{
	const void *addr = kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	if (addr)
	{
		const xpu_datum_operators *key_ops = sdesc->key_ops;

		if (key_ops->xpu_datum_heap_read(kcxt, addr, xdatum))
			return true;
	}
	xdatum->expr_ops = NULL;
	return true;
}

INLINE_FUNCTION(bool)
__gpusort_load_pminmax_int64(kern_context *kcxt,
							 const kern_sortkey_desc *sdesc,
							 const kern_data_store *kds_final,
							 const kern_tupitem *titem,
							 xpu_datum_t *__xdatum)
{
	const kagg_state__pminmax_int64_packed *x = (const kagg_state__pminmax_int64_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	assert(sdesc->key_ops == &xpu_int8_ops);
	if (x && (x->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
	{
		xpu_int8_t *xdatum = (xpu_int8_t *)__xdatum;
		xdatum->expr_ops = &xpu_int8_ops;
		xdatum->value = x->value;
	}
	else
	{
		__xdatum->expr_ops = NULL;
	}
	return true;

}

INLINE_FUNCTION(bool)
__gpusort_load_pminmax_fp64(kern_context *kcxt,
							const kern_sortkey_desc *sdesc,
							const kern_data_store *kds_final,
							const kern_tupitem *titem,
							xpu_datum_t *__xdatum)
{
	const kagg_state__pminmax_fp64_packed *x = (const kagg_state__pminmax_fp64_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	assert(sdesc->key_ops == &xpu_float8_ops);
	if (x && (x->attrs & __PAGG_MINMAX_ATTRS__VALID) != 0)
	{
		xpu_float8_t *xdatum = (xpu_float8_t *)__xdatum;
		xdatum->expr_ops = &xpu_float8_ops;
		xdatum->value = x->value;
	}
	else
	{
		__xdatum->expr_ops = NULL;
	}
	return true;
}

INLINE_FUNCTION(bool)
__gpusort_load_psum_int64(kern_context *kcxt,
						  const kern_sortkey_desc *sdesc,
						  const kern_data_store *kds_final,
						  const kern_tupitem *titem,
						  xpu_datum_t *__xdatum)
{
	const kagg_state__psum_int_packed *x = (const kagg_state__psum_int_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	assert(sdesc->key_ops == &xpu_int8_ops);
	if (x && x->nitems > 0)
	{
		xpu_int8_t *xdatum = (xpu_int8_t *)__xdatum;
		xdatum->expr_ops = &xpu_int8_ops;
		xdatum->value = x->sum;
	}
	else
	{
		__xdatum->expr_ops = NULL;
	}
	return true;
}

INLINE_FUNCTION(bool)
__gpusort_load_psum_fp64(kern_context *kcxt,
						 const kern_sortkey_desc *sdesc,
						 const kern_data_store *kds_final,
						 const kern_tupitem *titem,
						 xpu_datum_t *__xdatum)
{
	const kagg_state__psum_fp_packed *x = (const kagg_state__psum_fp_packed *)
		kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	assert(sdesc->key_ops == &xpu_float8_ops);
	if (x && x->nitems > 0)
	{
		xpu_float8_t *xdatum = (xpu_float8_t *)__xdatum;
		xdatum->expr_ops = &xpu_float8_ops;
		xdatum->value = x->sum;
	}
	else
	{
		__xdatum->expr_ops = NULL;
	}
	return true;
}

INLINE_FUNCTION(bool)
__gpusort_load_psum_numeric(kern_context *kcxt,
							const kern_sortkey_desc *sdesc,
							const kern_data_store *kds_final,
							const kern_tupitem *titem,
							xpu_datum_t *__xdatum)
{
	const kagg_state__psum_numeric_packed *x = (const kagg_state__psum_numeric_packed *)
        kern_fetch_minimal_tuple_attr(kds_final, titem, sdesc->src_anum);
	assert(sdesc->key_ops == &xpu_numeric_ops);
	if (x && x->nitems > 0)
	{
		xpu_numeric_t *xdatum = (xpu_numeric_t *)__xdatum;
		int		special = (x->attrs & __PAGG_NUMERIC_ATTRS__MASK);

		if (special == 0)
		{
			xdatum->kind = XPU_NUMERIC_KIND__VALID;
			xdatum->weight = (int16_t)(x->attrs & __PAGG_NUMERIC_ATTRS__WEIGHT);
			xdatum->u.value = __fetch_int128_packed(&x->sum);
		}
		else if (special == __PAGG_NUMERIC_ATTRS__PINF)
			xdatum->kind = XPU_NUMERIC_KIND__POS_INF;
		else if (special == __PAGG_NUMERIC_ATTRS__NINF)
			xdatum->kind = XPU_NUMERIC_KIND__NEG_INF;
		else
			xdatum->kind = XPU_NUMERIC_KIND__NAN;
		xdatum->expr_ops = &xpu_numeric_ops;
	}
	else
	{
		__xdatum->expr_ops = NULL;
	}
	return true;
}

INLINE_FUNCTION(bool)
__gpusort_load_precomp_fp64(kern_context *kcxt,
							const kern_sortkey_desc *sdesc,
							const kern_data_store *kds_final,
							const kern_tupitem *titem,
							xpu_datum_t *__xdatum)
{
	const char *addr = ((char *)titem + titem->t_len + sdesc->buf_offset);
	bool		notnull = *addr++;

	assert(sdesc->key_ops == &xpu_float8_ops);
	if (notnull)
	{
		xpu_float8_t *xdatum = (xpu_float8_t *)__xdatum;
		xdatum->expr_ops = &xpu_float8_ops;
		memcpy(&xdatum->value, addr, sizeof(float8_t));
	}
	else
	{
		__xdatum->expr_ops = NULL;
	}
	return true;
}

/*
 * kern_windowrank_prep_hash
 */
KERNEL_FUNCTION(void)
kern_windowrank_prep_hash(kern_session_info *session,
						  kern_gputask *kgtask,
						  kern_data_store *kds_final,
						  uint32_t *partition_hash_array)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	uint32_t	   *orderby_hash_array = partition_hash_array + kds_final->nitems;
	kern_context   *kcxt;
	uint32_t		index, sz = 0;
	xpu_datum_t	   *xdatum;

	/* save the GPU-Task specific read-only properties */
	if (get_local_id() == 0)
	{
		stromTaskProp__cuda_dindex        = kgtask->cuda_dindex;
		stromTaskProp__cuda_stack_limit   = kgtask->cuda_stack_limit;
		stromTaskProp__partition_divisor  = kgtask->partition_divisor;
		stromTaskProp__partition_reminder = kgtask->partition_reminder;
	}
	/* setup execution context */
	INIT_KERNEL_CONTEXT(kcxt, session, NULL);
	/* allocation of xdatum */
	for (int j=0; j < sort_kexp->u.sort.nkeys; j++)
	{
		const kern_sortkey_desc *sdesc = &sort_kexp->u.sort.desc[j];

		sz = Max(sdesc->key_ops->xpu_type_sizeof, sz);
	}
	xdatum = (xpu_datum_t *)alloca(sz);

	assert(sort_kexp->u.sort.window_partby_nkeys > 0);
	assert(sort_kexp->u.sort.window_orderby_nkeys > 0);
	for (index = get_global_id();
		 index < kds_final->nitems;
		 index += get_global_size())
	{
		kern_tupitem *titem = KDS_GET_TUPITEM(kds_final, index);
		uint32_t	hash = 0;

		for (int anum=1; anum <= sort_kexp->u.sort.nkeys; anum++)
		{
			const kern_sortkey_desc *sdesc = &sort_kexp->u.sort.desc[anum-1];
			const xpu_datum_operators  *key_ops = sdesc->key_ops;
			uint32_t	__hash;

			switch (sdesc->kind)
			{
				case KSORT_KEY_KIND__VREF:
					if (!__gpusort_load_rawkey(kcxt,
											   sdesc,
											   kds_final,
											   titem,
											   xdatum))
						goto bailout;
					break;
				case KSORT_KEY_KIND__PMINMAX_INT64:
					if (!__gpusort_load_pminmax_int64(kcxt,
													  sdesc,
													  kds_final,
													  titem,
													  xdatum))
						goto bailout;
					break;
				case KSORT_KEY_KIND__PMINMAX_FP64:
					if (!__gpusort_load_pminmax_fp64(kcxt,
													 sdesc,
													 kds_final,
													 titem,
													 xdatum))
						goto bailout;
					break;
				case KSORT_KEY_KIND__PSUM_INT64:
					if (!__gpusort_load_psum_int64(kcxt,
												   sdesc,
												   kds_final,
												   titem,
												   xdatum))
						goto bailout;
					break;
				case KSORT_KEY_KIND__PSUM_FP64:
					if (!__gpusort_load_psum_fp64(kcxt,
												  sdesc,
												  kds_final,
												  titem,
												  xdatum))
						goto bailout;
					break;
				case KSORT_KEY_KIND__PSUM_NUMERIC:
					if (!__gpusort_load_psum_numeric(kcxt,
													 sdesc,
													 kds_final,
													 titem,
													 xdatum))
                        goto bailout;
					break;
				case KSORT_KEY_KIND__PAVG_INT64:
				case KSORT_KEY_KIND__PAVG_FP64:
				case KSORT_KEY_KIND__PAVG_NUMERIC:
				case KSORT_KEY_KIND__PVARIANCE_SAMP:
				case KSORT_KEY_KIND__PVARIANCE_POP:
				case KSORT_KEY_KIND__PCOVAR_CORR:
				case KSORT_KEY_KIND__PCOVAR_SAMP:
				case KSORT_KEY_KIND__PCOVAR_POP:
				case KSORT_KEY_KIND__PCOVAR_AVGX:
				case KSORT_KEY_KIND__PCOVAR_AVGY:
				case KSORT_KEY_KIND__PCOVAR_COUNT:
				case KSORT_KEY_KIND__PCOVAR_INTERCEPT:
				case KSORT_KEY_KIND__PCOVAR_REGR_R2:
				case KSORT_KEY_KIND__PCOVAR_REGR_SLOPE:
				case KSORT_KEY_KIND__PCOVAR_REGR_SXX:
				case KSORT_KEY_KIND__PCOVAR_REGR_SXY:
				case KSORT_KEY_KIND__PCOVAR_REGR_SYY:
					if (!__gpusort_load_precomp_fp64(kcxt,
													 sdesc,
													 kds_final,
													 titem,
													 xdatum))
						goto bailout;
					break;
				default:
					STROM_ELOG(kcxt, "unknown sorting key kind");
					goto bailout;
			}
			if (!key_ops->xpu_datum_hash(kcxt, &__hash, xdatum))
				goto bailout;
			hash = ((hash >> 27) | (hash << 27)) ^ __hash;
			if (anum == sort_kexp->u.sort.window_partby_nkeys)
				partition_hash_array[index] = hash;
			if (anum == (sort_kexp->u.sort.window_partby_nkeys +
						 sort_kexp->u.sort.window_orderby_nkeys))
			{
				orderby_hash_array[index] = hash;
				break;
			}
		}
	}
bailout:
	STROM_WRITEBACK_ERROR_STATUS(&kgtask->kerror, kcxt);
}

/*
 * kern_windowrank_finalize
 */
KERNEL_FUNCTION(void)
kern_windowrank_finalize(kern_data_store *kds_final,
						 uint64_t old_length,
						 uint32_t old_nitems,
						 const uint32_t *results_array,
						 const uint64_t *windowrank_row_index)
{
	uint64_t	__nrooms = GPUSORT_WINDOWRANK_RESULTS_NROOMS(old_nitems);
	uint32_t	new_nitems = (__nrooms > 0 ? results_array[__nrooms-1] : 0);
	uint32_t	base;
	__shared__ uint64_t base_usage;

	for (base = get_global_base();
		 base < old_nitems;
		 base += get_global_size())
	{
		const kern_tupitem *titem = NULL;
		uint32_t	index = base + get_local_id();
		uint32_t	tupsz = 0;
		uint64_t	offset;
		uint64_t	total_sz;

		if (index < old_nitems && windowrank_row_index[index] != 0)
		{
			titem = (const kern_tupitem *)
				((char *)kds_final
				 + old_length
				 - windowrank_row_index[index]);
			tupsz = MAXALIGN(titem->t_len);
		}
		/* allocation of the destination buffer */
		offset = pgstrom_stair_sum_uint64(tupsz, &total_sz);
		if (get_local_id() == 0)
			base_usage = __atomic_add_uint64(&kds_final->usage,  total_sz);
		__syncthreads();
		/* put tuples on the destination */
		offset += base_usage;
		if (tupsz > 0)
		{
			kern_tupitem   *__titem = (kern_tupitem *)
				((char *)kds_final + kds_final->length - offset);
			uint32_t		__index = results_array[index] - 1;

			assert(__index < new_nitems);
			memcpy(__titem, titem, titem->t_len);
			KERN_TUPITEM_SET_ROWID(__titem, __index);
			__threadfence();
			KDS_GET_ROWINDEX(kds_final)[__index] = ((char *)kds_final
													+ kds_final->length
													- (char *)__titem);
		}
		__syncthreads();
	}
	if (get_global_id() == 0)
		kds_final->nitems = new_nitems;
}

/*
 * Simple GPU-Sort + LIMIT clause
 */
KERNEL_FUNCTION(void)
kern_buffer_simple_limit(kern_data_store *kds_final, uint64_t old_length)
{
	uint64_t   *row_index = KDS_GET_ROWINDEX(kds_final);
	uint32_t	base;
	__shared__ uint64_t base_usage;

	assert(kds_final->format == KDS_FORMAT_ROW ||
		   kds_final->format == KDS_FORMAT_HASH);
	for (base = get_global_base();
		 base < kds_final->nitems;
		 base += get_global_size())
	{
		const kern_tupitem *titem = NULL;
		uint32_t	index = base + get_local_id();
		uint32_t	tupsz = 0;
		uint64_t	offset;
		uint64_t	total_sz;

		if (index < kds_final->nitems)
		{
			// XXX - must not use KDS_GET_TUPITEM() because kds_final->length
			//       is already truncated.
			assert(row_index[index] != 0);
			titem = (const kern_tupitem *)
				((char *)kds_final + old_length - row_index[index]);
			tupsz = MAXALIGN(titem->t_len);
		}
		/* allocation of the destination buffer */
		offset = pgstrom_stair_sum_uint64(tupsz, &total_sz);
		if (get_local_id() == 0)
			base_usage = __atomic_add_uint64(&kds_final->usage,  total_sz);
		__syncthreads();
		/* put tuples on the destination */
		offset += base_usage;
		if (tupsz > 0)
		{
			kern_tupitem   *__titem = (kern_tupitem *)
				((char *)kds_final + kds_final->length - offset);
			memcpy(__titem, titem, titem->t_len);
			KERN_TUPITEM_SET_ROWID(__titem, index);
			__threadfence();
			row_index[index] = ((char *)kds_final
								+ kds_final->length
								- (char *)__titem);
		}
		__syncthreads();
	}
}

/*
 * GPU Buffer simple reconstruction (ROW-format with PARTITION)
 */
KERNEL_FUNCTION(void)
kern_gpusort_partition_buffer(kern_session_info *session,
							  kern_gputask *kgtask,
							  kern_data_store *kds_dst,
							  kern_data_store *kds_src)
{
	const kern_expression *sort_kexp = SESSION_KEXP_GPUSORT_KEYDESC(session);
	kern_context   *kcxt;
	uint32_t		base;
	xpu_datum_t	   *xdatum = NULL;
	__shared__ uint32_t base_rowid;
	__shared__ uint64_t base_usage;

	/* save the GPU-Task specific read-only properties */
	if (get_local_id() == 0)
	{
		stromTaskProp__cuda_dindex        = kgtask->cuda_dindex;
		stromTaskProp__cuda_stack_limit   = kgtask->cuda_stack_limit;
		stromTaskProp__partition_divisor  = kgtask->partition_divisor;
		stromTaskProp__partition_reminder = kgtask->partition_reminder;
	}
	/* setup execution context */
	INIT_KERNEL_CONTEXT(kcxt, session, NULL);
	__syncthreads();

	assert((kds_src->format == KDS_FORMAT_ROW ||
			kds_src->format == KDS_FORMAT_HASH) &&
		   (kds_dst->format == KDS_FORMAT_ROW));
	/* allocation of xdatum */
	if (sort_kexp && stromTaskProp__partition_divisor > 0)
	{
		int		sz = sizeof(xpu_datum_t);

		for (int j=0; j < sort_kexp->u.sort.nkeys; j++)
		{
			const kern_sortkey_desc *sdesc = &sort_kexp->u.sort.desc[j];

			sz = Max(sdesc->key_ops->xpu_type_sizeof, sz);
		}
		xdatum = (xpu_datum_t *)alloca(sz);
	}

	for (base = get_global_base();
		 base < kds_src->nitems;
		 base += get_global_size())
	{
		kern_tupitem *titem = NULL;
		uint32_t	index = base + get_local_id();
		uint32_t	tupsz = 0;
		uint32_t	row_id;
		uint32_t	count;
		uint64_t	offset;
		uint64_t	total_sz;

		if (index < kds_src->nitems)
		{
			titem = KDS_GET_TUPITEM(kds_src, index);
			if (stromTaskProp__partition_divisor == 0)
			{
				/*
				 * Although not currently used, if the divisor is zero, it is
				 * considered as a simple reconstruction operation without
				 * partitioning.
				 */
				tupsz = MAXALIGN(titem->t_len);
			}
			else if (stromTaskProp__partition_reminder == 0)
			{
				/*
				 * When partitioning the buffer, the GPU kernel is called in order,
				 * starting with the remainder 0 and ending with (divisor-1).
				 * At this time, the sort key used for the window function is hashed
				 * and partitioning is performed by this value, but this only needs
				 * to be calculated once at the beginning; in the case of remainders
				 * 1, 2, ..., the value is also written to the kds_src side so that
				 * it is sufficient to refer to kern_tupitem->hash.
				 */
				uint32_t	hash = 0;

				for (int anum=1; anum <= sort_kexp->u.sort.nkeys; anum++)
				{
					const kern_sortkey_desc *sdesc = &sort_kexp->u.sort.desc[anum-1];
					const xpu_datum_operators  *key_ops = sdesc->key_ops;
					uint32_t		__hash;

					assert(sdesc->kind == KSORT_KEY_KIND__VREF);
					if (sdesc->kind != KSORT_KEY_KIND__VREF)
					{
						STROM_ELOG(kcxt, "unexpected sorting key kind");
						break;
					}
					if (!__gpusort_load_rawkey(kcxt,
											   sdesc,
											   kds_src,
											   titem,
											   xdatum) ||
						!key_ops->xpu_datum_hash(kcxt,
												 &__hash,
												 xdatum))
						break;
					hash = ((hash >> 27) | (hash << 27)) ^ __hash;
					if (anum == sort_kexp->u.sort.window_partby_nkeys)
					{
						titem->hash = hash;
						if ((hash % stromTaskProp__partition_divisor) == stromTaskProp__partition_reminder)
							tupsz = MAXALIGN(titem->t_len);
						break;
					}
				}
			}
			else if ((titem->hash % stromTaskProp__partition_divisor) == stromTaskProp__partition_reminder)
			{
				tupsz = MAXALIGN(titem->t_len);
			}
		}
		/* allocation of the destination buffer */
		row_id = pgstrom_stair_sum_binary(tupsz > 0, &count);
		offset = pgstrom_stair_sum_uint64(tupsz, &total_sz);
		if (get_local_id() == 0)
		{
			base_rowid = __atomic_add_uint32(&kds_dst->nitems, count);
			base_usage = __atomic_add_uint64(&kds_dst->usage,  total_sz);
		}
		__syncthreads();
		/* put tuples on the destination */
		row_id += base_rowid;
		offset += base_usage;
		if (tupsz > 0)
		{
			kern_tupitem   *__titem = (kern_tupitem *)
				((char *)kds_dst + kds_dst->length - offset);
			memcpy(__titem, titem, titem->t_len);
			KERN_TUPITEM_SET_ROWID(__titem, row_id);
			__threadfence();
			KDS_GET_ROWINDEX(kds_dst)[row_id] = offset;
		}
		__syncthreads();
	}
}

/*
 * GPU Buffer simple reconstruction (ROW-format by simple consolidation)
 */
KERNEL_FUNCTION(void)
kern_gpusort_consolidate_buffer(kern_data_store *kds_dst,
								const kern_data_store *__restrict__ kds_src)
{
	__shared__ uint32_t base_rowid;
	__shared__ uint64_t base_usage;
	uint32_t	base;

	assert(kds_dst->format == KDS_FORMAT_ROW &&
		   kds_src->format == KDS_FORMAT_ROW);
	for (base = get_global_base();
		 base < kds_src->nitems;
		 base += get_global_size())
	{
		const kern_tupitem *titem = NULL;
		uint32_t	index = base + get_local_id();
		uint32_t	tupsz = 0;
		uint32_t	row_id;
		uint32_t	count;
		uint64_t	offset;
		uint64_t	total_sz;

		if (index < kds_src->nitems)
		{
			titem = KDS_GET_TUPITEM(kds_src, index);
			tupsz = MAXALIGN(titem->t_len);
		}
		/* allocation of the destination buffer */
		row_id = pgstrom_stair_sum_binary(tupsz > 0, &count);
		offset = pgstrom_stair_sum_uint64(tupsz, &total_sz);
		if (get_local_id() == 0)
		{
			base_rowid = __atomic_add_uint32(&kds_dst->nitems, count);
			base_usage = __atomic_add_uint64(&kds_dst->usage,  total_sz);
		}
		__syncthreads();
		/* put tuples on the destination */
		row_id += base_rowid;
		offset += base_usage;
		if (tupsz > 0)
		{
			kern_tupitem   *__titem = (kern_tupitem *)
				((char *)kds_dst + kds_dst->length - offset);
			memcpy(__titem, titem, titem->t_len);
			KERN_TUPITEM_SET_ROWID(__titem, row_id);
			__threadfence();
			KDS_GET_ROWINDEX(kds_dst)[row_id] = offset;
		}
		__syncthreads();
	}
}
